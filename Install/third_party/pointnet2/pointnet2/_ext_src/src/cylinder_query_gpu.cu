#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates.
// 
// This source code is licensed under the MIT license found in the
// LICENSE file in the root directory of this source tree.

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"

// input: new_xyz(b, m, 3) xyz(b, n, 3) rot_c2w(b, m, 9)
// output: idx(b, m, nsample)
__global__ void query_cylinder_point_kernel(int b, int n, int m, float radius, float hmin, float hmax,
                                        int nsample,
                                        const float *__restrict__ new_xyz,
                                        const float *__restrict__ xyz,
                                        const float *__restrict__ rot,
                                        int *__restrict__ idx) {
  int batch_index = blockIdx.x;
  xyz += batch_index * n * 3;
  new_xyz += batch_index * m * 3;
  rot += batch_index * m * 9;
  idx += m * nsample * batch_index;

  int index = threadIdx.x;
  int stride = blockDim.x;

  float radius2 = radius * radius;
  for (int j = index; j < m; j += stride) {
    float new_x = new_xyz[j * 3 + 0];
    float new_y = new_xyz[j * 3 + 1];
    float new_z = new_xyz[j * 3 + 2];
    float r0 = rot[j * 9 + 0];
    float r1 = rot[j * 9 + 1];
    float r2 = rot[j * 9 + 2];
    float r3 = rot[j * 9 + 3];
    float r4 = rot[j * 9 + 4];
    float r5 = rot[j * 9 + 5];
    float r6 = rot[j * 9 + 6];
    float r7 = rot[j * 9 + 7];
    float r8 = rot[j * 9 + 8];
    for (int k = 0, cnt = 0; k < n && cnt < nsample; ++k) {
      float x = xyz[k * 3 + 0] - new_x;
      float y = xyz[k * 3 + 1] - new_y;
      float z = xyz[k * 3 + 2] - new_z;
      float x_rot = r0 * x + r3 * y + r6 * z;
      float y_rot = r1 * x + r4 * y + r7 * z;
      float z_rot = r2 * x + r5 * y + r8 * z;
      float d2 = y_rot * y_rot + z_rot * z_rot;
      if (d2 < radius2 && x_rot > hmin && x_rot < hmax) {
        if (cnt == 0) {
          for (int l = 0; l < nsample; ++l) {
            idx[j * nsample + l] = k;
          }
        }
        idx[j * nsample + cnt] = k;
        ++cnt;
      }
    }
  }
}

void query_cylinder_point_kernel_wrapper(int b, int n, int m, float radius, float hmin, float hmax,
                                     int nsample, const float *new_xyz,
                                     const float *xyz, const float *rot, int *idx) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  query_cylinder_point_kernel<<<b, opt_n_threads(m), 0, stream>>>(
      b, n, m, radius, hmin, hmax, nsample, new_xyz, xyz, rot, idx);

  CUDA_CHECK_ERRORS();
}
